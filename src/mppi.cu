
#include <hip/hip_runtime.h>
// cuda code for MPPI

#define SAMPLE_COUNT 1024
#define HORIZON 20

#define CONTROL_DIM 2
#define STATE_DIM 4
#define TEMPERATURE 1

__global__
void evaluate_control_sequence(float *out_cost, float *x0, float *in_control, float *in_epsilon){
  // get global thread id
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id>=SAMPLE_COUNT){
    return;
  }
  printf("id = %d\n",id);
  float x[STATE_DIM];
  // copy to local state
  // TODO error
  for (int i=0; i<STATE_DIM; i++){
    x[i] = *(x0 + i);
  }
  //printf("id = %d, x0, %.2f, %.2f, %.2f, %.2f \n",id,x0[0],x0[1],x0[2],x0[3]);

  // prepare constants
  const float m1 = 1;
  const float m2 = 1;
  const float k1 = 1;
  const float k2 = 1;
  const float c1 = 1.4;
  const float c2 = 1.4;
  const float dt = 0.1;

  // initialize cost
  //out_cost[id] = 0;
  float cost = 0;
  // run simulation
  for (int i=0; i<HORIZON; i++){
    float u0 = in_control[id*HORIZON*CONTROL_DIM + i*CONTROL_DIM];
    float u1 = in_control[id*HORIZON*CONTROL_DIM + i*CONTROL_DIM + 1];

    // step forward dynamics, update state x
    float x1 = x[0];
    float dx1 = x[1];
    float x2 = x[2];
    float dx2 = x[3];

    float ddx1 = -(k1*x1 + c1*dx1 + k2*(x1-x2) + c2*(dx1-dx2)-u0)/m1;
    float ddx2 = -(k2*(x2-x1) + c2*(dx2-dx1)-u1)/m2;

    //printf("id = %d, ddx1=%.2f, ddx2=%.2f \n",id,ddx1,ddx2);
    //float temp=-(k1*x1 + c1*dx1 + k2*(x1-x2) + c2*(dx1-dx2)-u0)/m1;
    x1 += dx1*dt;
    dx1 += ddx1*dt;
    x2 += dx2*dt;
    dx2 += ddx2*dt;
    //printf("id = %d, step = %d cost = %.2f, x= %.2f, %.2f, %.2f, %.2f \n",id,i,cost,x1,dx1,x2,dx2);

    x[0] = x1;
    x[1] = dx1;
    x[2] = x2;
    x[3] = dx2;

    // evaluate cost, update cost
    cost = cost + (x[0]-1)*(x[0]-1)*1.0 + x[1]*x[1]*0.01 + (x[2]-3)*(x[2]-3)*1.0 + x[3]*x[3]*0.01;
    cost = cost + u0*in_epsilon[id*HORIZON*CONTROL_DIM + i*CONTROL_DIM] + u1*in_epsilon[id*HORIZON*CONTROL_DIM + i*CONTROL_DIM + 1];

  }
  out_cost[id] = cost;


}
